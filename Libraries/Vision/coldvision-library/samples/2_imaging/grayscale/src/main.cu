#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Nov 19, 2015
 *      Author: claudiu
 */

#include <iostream>
#include <string>
#include <stdio.h>
#include "utils.h"
#include "timer.h"
#include "grayscale.cuh"

void processUsingCuda(std::string input_file, std::string output_file);
void processUsingOpenCV(std::string input_file, std::string output_file);
void processUsingCpu(std::string input_file, std::string output_file);

int main(int argc, char **argv) {

	std::string input_file;
	std::string output_cuda_file  = "../data/output_cuda.png";
	std::string output_opencv_file = "../data/output_opencv.png";
	std::string output_cpu_file = "../data/output_cpu.png";

	// used for the allowed error between different implementations
	bool useEpsCheck = true; // set true to enable perPixelError and globalError
	double perPixelError = 3;
	double globalError   = 10;

	switch (argc)
	{
	case 2:
		input_file = std::string(argv[1]);
		break;
	case 3:
		input_file  = std::string(argv[1]);
		output_cuda_file = std::string(argv[2]);
		break;
	case 4:
		input_file  = std::string(argv[1]);
		output_cuda_file = std::string(argv[2]);
		output_opencv_file = std::string(argv[3]);
		break;
	case 5:
		input_file  = std::string(argv[1]);
		output_cuda_file = std::string(argv[2]);
		output_opencv_file = std::string(argv[3]);
		output_cpu_file = std::string(argv[4]);
		break;
	case 7:
		useEpsCheck=true;
		input_file  = std::string(argv[1]);
		output_cuda_file = std::string(argv[2]);
		output_opencv_file = std::string(argv[3]);
		output_cpu_file = std::string(argv[4]);
		perPixelError = atof(argv[5]);
		globalError   = atof(argv[6]);
		break;
	default:
		std::cerr << "Usage: ./grayscale input_file [output_cuda] [output_opencv] [output_opencv] [output_cpu] [globalError]" << std::endl;
		exit(1);
	}

	for (int i=0; i<10; ++i) {
		processUsingOpenCV(input_file, output_opencv_file);
		processUsingCuda(input_file, output_cuda_file);
		processUsingCpu(input_file, output_cpu_file);
	}

	// check if the generated images are the same
	compareImages(output_cpu_file, output_cuda_file, useEpsCheck, perPixelError, globalError);

	cleanupCuda();

	return 0;
}

void processUsingCuda(std::string input_file, std::string output_file) {
	// pointers to images in CPU's memory (h_) and GPU's memory (d_)
	uchar4        *h_rgbaImage, *d_rgbaImage;
	unsigned char *h_greyImage, *d_greyImage;

	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

	GpuTimer timer;
	timer.Start();
	// here is where the conversion actually happens
	rgbaToGreyscaleCuda(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
	timer.Stop();
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	int err = printf("Implemented CUDA code ran in: %f msecs.\n", timer.Elapsed());

	if (err < 0) {
		//Couldn't print!
		std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
		exit(1);
	}

	size_t numPixels = numRows()*numCols();
	checkCudaErrors(hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));

	//check results and output the grey image
	postProcess(output_file, h_greyImage);
}

void processUsingOpenCV(std::string input_file, std::string output_file) {
	cv::Mat image;
	image = cv::imread(input_file.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << input_file << std::endl;
		exit(1);
	}

	GpuTimer timer;
	timer.Start();
	cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);  // CV_BGR2GRAY

	//allocate memory for the output
	imageGrey.create(image.rows, image.cols, CV_8UC1);
	timer.Stop();

	int err = printf("OpenCV code ran in: %f msecs.\n", timer.Elapsed());

	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}

	//output the image
	cv::imwrite(output_file.c_str(), imageGrey);
}

void processUsingCpu(std::string input_file, std::string output_file) {
	// pointers to images in CPU's memory (h_) and GPU's memory (d_)
	uchar4        *h_rgbaImage, *d_rgbaImage;
	unsigned char *h_greyImage, *d_greyImage;

	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

	GpuTimer timer;
	timer.Start();
	rgbaToGreyscaleCpu(h_rgbaImage, h_greyImage, numRows(), numCols());
	timer.Stop();

	int err = printf("Implemented CPU serial code ran in: %f msecs.\n", timer.Elapsed());

	if (err < 0) {
		//Couldn't print!
		std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
		exit(1);
	}

	//check results and output the grey image
	postProcess(output_file, h_greyImage);
}
