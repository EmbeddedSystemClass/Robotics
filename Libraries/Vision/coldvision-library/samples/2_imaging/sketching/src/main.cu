#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "opencv2/imgproc.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/videoio.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/cudaimgproc.hpp"
#include "opencv2/cudafilters.hpp" // cv::cuda::Filter
#include "opencv2/cudaarithm.hpp" // cv::cuda::abs or cv::cuda::addWeighted
#include "timer.h"

namespace cpu {
void processImage(std::string inputFile, std::string outputFile);
void processVideo(std::string inputFile, std::string outputFile);
}

namespace gpu {
void processImage(std::string inputFile, std::string outputFile);
void processVideo(std::string inputFile, std::string outputFile);
}

using namespace std;
using namespace cv;

int main(int argc, char **argv) {

	for (int i=1; i<=5; ++i) {
		string fileId = std::to_string(i);
		const string inputFile = argc >= 2 ? argv[1] : "../data/img_color_"+fileId+".jpg";
		const string outputFile = argc >= 3 ? argv[2] : "../data/out_img_color_"+fileId+"_sketching.jpg";
		cpu::processImage(inputFile, outputFile);
		gpu::processImage(inputFile, outputFile);
	}

	for (int i=1; i<=5; ++i) {
		string fileId = std::to_string(i);
		const string inputFile = argc >= 2 ? argv[1] : "../data/img_ir_"+fileId+".jpg";
		const string outputFile = argc >= 3 ? argv[2] : "../data/out_img_ir_"+fileId+"_sketching.jpg";
		cpu::processImage(inputFile, outputFile);
		gpu::processImage(inputFile, outputFile);
	}

	cpu::processVideo("../data/video_color_2.mp4", "../data/out_video_color_2.mp4");
	gpu::processVideo("../data/video_color_2.mp4", "../data/out_video_color_2.mp4");

	return 0;
}

namespace cpu {

/**
 * It reduces a value to a interval value (continue to discrete value).
 * For a value between [n, n+intervalSize), where n is a multiple of intervalSize, this method returns n.
 * Example 1 (val=11, intervalSize=4): return=8
 * Example 2 (val=27, intervalSize=8): return=24
 */
inline uchar reduceValue(const uchar value, const uchar intervalSize = 4) {
	return value / intervalSize * intervalSize;
}

/**
 * It reduces the image color palette from a total number 256*256*256 of unique colors to something smaller like 32*32*32.
 * @see http://stackoverflow.com/questions/5906693/how-to-reduce-the-number-of-colors-in-an-image-with-opencv
 */
void reduceColorPalette(const Mat& inputImage, Mat& outputImage, const uchar intervalSize = 4)
{
	uchar* inputPixelPtr = inputImage.data;
	uchar* outputPixelPtr = outputImage.data;

	if(inputImage.channels() == 3) { // color image
		for (int i = 0; i < inputImage.rows; i++) {
			for (int j = 0; j < inputImage.cols; j++) {
				const int pi = i*inputImage.cols*3 + j*3;
				outputPixelPtr[pi + 0] = reduceValue(inputPixelPtr[pi + 0]); // B
				outputPixelPtr[pi + 1] = reduceValue(inputPixelPtr[pi + 1]); // G
				outputPixelPtr[pi + 2] = reduceValue(inputPixelPtr[pi + 2]); // R
			}
		}

	} else if(inputImage.channels() == 1) { // grayscale image
		for (int i = 0; i < inputImage.rows; i++) {
			for (int j = 0; j < inputImage.cols; j++) {
				const int pi = i*inputImage.cols*3 + j*3;
				outputPixelPtr[pi] = reduceValue(inputPixelPtr[pi]); // gray
			}
		}

	} else { // not supported image format
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}
}

/**
 * This method makes histogram equalization of the pixel intensities.
 * For grayscale images the equalization is done directly using the values of the grayscale channel,
 * but color images the image is transformed to other color space than RGB which separates intensity values
 * from color components, color spaces such HSV/HLS, YUV or YCbCr.
 *
 * @see http://stackoverflow.com/questions/15007304/histogram-equalization-not-working-on-color-image-opencv
 */
void equalizeIntensity(const Mat& inputImage, Mat& outputImage)
{
	if(inputImage.channels() == 3) { // color image
		Mat ycrcb;
		cvtColor(inputImage, ycrcb, CV_BGR2HSV);
		vector<Mat> channels;
		split(ycrcb, channels);
		equalizeHist(channels[0], channels[0]);
		Mat result;
		merge(channels, ycrcb);
		cvtColor(ycrcb, outputImage, CV_HSV2BGR);

	} else if(inputImage.channels() == 1) { // grayscale image
		equalizeHist(inputImage, outputImage);

	} else { // not supported image format
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}
}

/**
 * This method computes the gradients of a grayscale image.
 * @param inputImage A grayscale image of type CV_8U
 * @param outputImage A grayscale image of type CV_8U
 */
void computeGradients(const Mat& inputImage, Mat& outputImage)
{
	if(inputImage.channels() == 1) { // grayscale
		// compute the gradients on both directions x and y
		Mat grad_x, grad_y;
		Mat abs_grad_x, abs_grad_y;
		int scale = 1;
		int delta = 0;
		int ddepth = CV_16S; // use 16 bits unsigned to avoid overflow

		Scharr( inputImage, grad_x, ddepth, 1, 0, scale, delta, BORDER_DEFAULT );
		//Sobel( input_gray, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
		convertScaleAbs( grad_x, abs_grad_x ); // CV_16S -> CV_8U

		Scharr( inputImage, grad_y, ddepth, 0, 1, scale, delta, BORDER_DEFAULT );
		//Sobel( input_gray, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );
		convertScaleAbs( grad_y, abs_grad_y ); // CV_16S -> // CV_16S -> CV_8U

		// create the output by adding the absolute gradient images of each x and y direction
		addWeighted( abs_grad_x, 0.5, abs_grad_y, 0.5, 0, outputImage );

	} else {
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}

}

/**
 * This method extracts the gradient image of a color or grayscale image.
 * @param inputImage A matrix of a color or grayscale image
 * @param outputImage The gradient matrix
 */
void processImage(const Mat& inputImage, Mat& outputImage)
{
	if (inputImage.channels() == 3) { // color image
		/// Apply Histogram Equalization
		//equalizeIntensity(inputImage, inputImage);

		// Reduce the maximum number of colors from 256*256*256 to a smaller number such 32*32*32
		//reduceNumberOfColors(inputImage, inputImage);

		// Blur the input image to remove the noise
		GaussianBlur(inputImage, inputImage, Size(9, 9), 0, 0, BORDER_DEFAULT);

		// Convert it to grayscale (CV_8UC3 -> CV_8UC1)
		Mat image_gray;
		cvtColor(inputImage, image_gray, COLOR_BGR2GRAY);

		// Compute the gradient image
		computeGradients(image_gray, image_gray);
		normalize(image_gray, outputImage, 0, 255, NORM_MINMAX, CV_8U);
		//threshold(outputImage, outputImage, 50, 255, THRESH_TOZERO);

		// invert the gradient image
		cv::subtract(cv::Scalar::all(255), outputImage, outputImage);

	} else if (inputImage.channels() == 1) { // grayscale image
		Mat image_gray = inputImage;
		computeGradients(image_gray, image_gray);
		normalize(image_gray, outputImage, 0, 255, NORM_MINMAX, CV_8U);
		//threshold(outputImage, outputImage, 50, 255, THRESH_TOZERO);

		// invert the gradient image
		cv::subtract(cv::Scalar::all(255), outputImage, outputImage);

	} else { // not supported image format
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}
}

/**
 * This method extracts the gradient image of a color or grayscale image.
 * The output image is saved as grayscale.
 */
void processImage(std::string inputFile, std::string outputFile) {
	printf("CPU::Processing image: %s ...\n", inputFile.c_str());

	// Read the file
	Mat inputImage = imread(inputFile, CV_LOAD_IMAGE_UNCHANGED);
	if (!inputImage.data) {
		printf("Could not open image file: %s\n", inputFile.c_str());
		return;
	}

	// Init the output image as grayscale with the same size as the input
	Mat outputImage (inputImage.size(), CV_8U);

	// Process the image
	GpuTimer timer;
	timer.Start();
	processImage(inputImage, outputImage);
	timer.Stop();

	printf("Method processImage() ran in: %f msecs, image size: %ux%u, msecs/pixel: %f .\n",
			timer.Elapsed(), inputImage.cols, inputImage.rows, timer.Elapsed()/(inputImage.rows*inputImage.cols));

	// Display the output image
	//imshow("Final Result", outputImage);

	// Wait until the user presses a key
	//waitKey(0);

	imwrite(outputFile, outputImage);
}

/**
 * This method processes each frame of a color or grayscale video and saves them in a gradient video.
 * The output video is saved in a color format even if each frame was grayscale.
 */
void processVideo(std::string inputFile, std::string outputFile) {
	printf("CPU::Processing video: %s ...\n", inputFile.c_str());

	// Read the file
	VideoCapture inputVideo (inputFile);
	if (!inputVideo.isOpened()) {
		printf("Could not open video file: %s\n", inputFile.c_str());
		return;
	}

	// Init the output video with the same properties as the input
	int fourcc = inputVideo.get(CAP_PROP_FOURCC);
	double fps = inputVideo.get(CAP_PROP_FPS);
	Size frameSize (inputVideo.get(CAP_PROP_FRAME_WIDTH), inputVideo.get(CAP_PROP_FRAME_HEIGHT));
	VideoWriter outputVideo(outputFile, fourcc, fps, frameSize, true);
	if (!outputVideo.isOpened()) {
		printf("Could not open the output video for write: %s\n", outputFile.c_str());
		return;
	}

	// Process the video, frame by frame
	GpuTimer timer;
	timer.Start();
	Mat frame, gradientFrame;
	while (inputVideo.read(frame)) {
		gradientFrame = Mat(frame.size(),CV_8U); // as grayscale
		processImage(frame, gradientFrame); // extract grayscale gradient from the color frame
		cvtColor(gradientFrame, frame, COLOR_GRAY2BGR); // convert grayscale gradient to color
		outputVideo.write(frame);
	}
	timer.Stop();

	printf("Method processVideo() ran in: %f msecs, video size: %ux%u, total frames: %f, msecs/pixel: %f .\n",
			timer.Elapsed(), frameSize.width, frameSize.height, inputVideo.get(CAP_PROP_FRAME_COUNT),
			timer.Elapsed()/(frameSize.height*frameSize.width*inputVideo.get(CAP_PROP_FRAME_COUNT)));

	outputVideo.release();
	inputVideo.release();
}

}

namespace gpu {

/**
 * This method makes histogram equalization of the pixel intensities.
 * For grayscale images the equalization is done directly using the values of the grayscale channel,
 * but color images the image is transformed to other color space than RGB which separates intensity values
 * from color components, color spaces such HSV/HLS, YUV or YCbCr.
 *
 * @see http://stackoverflow.com/questions/15007304/histogram-equalization-not-working-on-color-image-opencv
 */
void equalizeIntensity(const cv::cuda::GpuMat& inputImage, cv::cuda::GpuMat& outputImage)
{
	if(inputImage.channels() == 3) { // color image
		cv::cuda::GpuMat ycrcb;
		cv::cuda::cvtColor(inputImage, ycrcb, CV_BGR2HSV);
		vector<cv::cuda::GpuMat> channels;
		cv::cuda::split(ycrcb, channels);
		cv::cuda::equalizeHist(channels[0], channels[0]);
		cv::cuda::merge(channels, ycrcb);
		cv::cuda::cvtColor(ycrcb, outputImage, CV_HSV2BGR);
		ycrcb.release();

	} else if(inputImage.channels() == 1) { // grayscale image
		cv::cuda::equalizeHist(inputImage, outputImage);

	} else { // not supported image format
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}
}

/**
 * This method computes the gradients of a grayscale image.
 * @param inputImage A grayscale image of type CV_8U
 * @param outputImage A grayscale image of type CV_8U
 */
void computeGradients(const cv::cuda::GpuMat& inputImage, cv::cuda::GpuMat& outputImage)
{
	if(inputImage.channels() == 1) { // grayscale

		// compute the gradients on both directions x and y
		cv::cuda::GpuMat grad_x, grad_y;
		cv::cuda::GpuMat abs_grad_x, abs_grad_y;
		int scale = 1;
		int ddepth = CV_16S; // use 16 bits unsigned to avoid overflow
		Ptr<cv::cuda::Filter> filter;

		// gradient x direction
		//filter = cv::cuda::createSobelFilter(inputImage.type(), ddepth, 1, 0, 3, scale, BORDER_DEFAULT);
		filter = cv::cuda::createScharrFilter(inputImage.type(), ddepth, 1, 0, scale, BORDER_DEFAULT);
		filter->apply(inputImage, grad_x);
		cv::cuda::abs(grad_x, grad_x);
		grad_x.convertTo(abs_grad_x, CV_8UC1); // CV_16S -> CV_8U

		// gradient y direction
		//filter = cv::cuda::createSobelFilter(inputImage.type(), ddepth, 0, 1, 3, scale, BORDER_DEFAULT);
		filter = cv::cuda::createScharrFilter(inputImage.type(), ddepth, 0, 1, scale, BORDER_DEFAULT);
		filter->apply(inputImage, grad_y);
		cv::cuda::abs(grad_y, grad_y);
		grad_y.convertTo(abs_grad_y, CV_8UC1); // CV_16S -> CV_8U

		// create the output by adding the absolute gradient images of each x and y direction
		cv::cuda::addWeighted(abs_grad_x, 0.5, abs_grad_y, 0.5, 0, outputImage);

		// release GPU memory
		grad_x.release();
		grad_y.release();
		abs_grad_x.release();
		abs_grad_y.release();

	} else {
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}
}

/**
 * This method extracts the inverted gradient image of a color or grayscale image.
 * @param inputImage A matrix of a color or grayscale image
 * @param outputImage The inverted gradient matrix
 */
void processImage(const cv::cuda::GpuMat& inputImage, cv::cuda::GpuMat& outputImage)
{
	if (inputImage.channels() == 3) { // color image

		// All transformations are done on the output image
		outputImage = inputImage;

		/// Apply Histogram Equalization
		//equalizeIntensity(outputImage, outputImage);

		// Reduce the maximum number of colors from 256*256*256 to a smaller number such 32*32*32
		//reduceNumberOfColors(inputImage, inputImage);

		// Blur the input image to remove the noise
		Ptr<cv::cuda::Filter> filter = cv::cuda::createGaussianFilter(outputImage.type(), outputImage.type(), Size(9,9), 0);
		filter->apply(outputImage, outputImage);

		// Convert it to grayscale (CV_8UC3 -> CV_8UC1)
		cv::cuda::cvtColor(outputImage, outputImage, COLOR_RGB2GRAY);

		// Compute the gradient image
		computeGradients(outputImage, outputImage);
		//normalize(outputImage, outputImage, 0, 255, NORM_MINMAX, CV_8U);
		//cv::cuda::threshold(outputImage, outputImage, 50, 255, THRESH_TOZERO);

		// invert the gradient image
		cv::cuda::subtract(cv::Scalar::all(255), outputImage, outputImage);

	} else if (inputImage.channels() == 1) { // grayscale image
		computeGradients(inputImage, outputImage);
		//normalize(image_gray, outputImage, 0, 255, NORM_MINMAX, CV_8U);
		//threshold(outputImage, outputImage, 50, 255, THRESH_TOZERO);

		// invert the gradient image
		cv::cuda::subtract(cv::Scalar::all(255), outputImage, outputImage);

	} else { // not supported image format
		printf("Image type not supported.\n");
		outputImage = inputImage;
	}
}

/**
 * This method extracts the gradient image of a color or grayscale image.
 * The output image is saved as grayscale.
 */
void processImage(std::string inputFile, std::string outputFile) {
	printf("GPU::Processing image: %s ...\n", inputFile.c_str());

	// Read the file
	Mat inputImage = imread(inputFile, CV_LOAD_IMAGE_UNCHANGED);
	if (!inputImage.data) {
		printf("Could not open image file: %s\n", inputFile.c_str());
		return;
	}

	// Init the output image as grayscale with the same size as the input
	Mat outputImage (inputImage.size(), CV_8U);

	// copy the input image from CPU to GPU memory
	cuda::GpuMat gpuInputImage = cuda::GpuMat(inputImage);
	cuda::GpuMat gpuOutputImage = cuda::GpuMat(outputImage);

	// Process the image
	GpuTimer timer;
	timer.Start();
	processImage(gpuInputImage, gpuOutputImage);
	timer.Stop();

	printf("Method processImage() ran in: %f msecs, image size: %ux%u, msecs/pixel: %f .\n",
			timer.Elapsed(), inputImage.cols, inputImage.rows, timer.Elapsed()/(inputImage.rows*inputImage.cols));

	// copy the result gradient from GPU to CPU and release GPU memory
	gpuOutputImage.download(outputImage);

	gpuInputImage.release();
	gpuOutputImage.release();

	// Display the output image
	//imshow("Final Result", outputImage);

	// Wait until the user presses a key
	//waitKey(0);

	imwrite(outputFile, outputImage);
}

/**
 * This method processes each frame of a color or grayscale video and saves them in a gradient video.
 * The output video is saved in a color format even if each frame was grayscale.
 */
void processVideo(std::string inputFile, std::string outputFile) {
	printf("GPU::Processing video: %s ...\n", inputFile.c_str());

	// Read the file
	VideoCapture inputVideo (inputFile);
	if (!inputVideo.isOpened()) {
		printf("Could not open video file: %s\n", inputFile.c_str());
		return;
	}

	// Init the output video with the same properties as the input
	int fourcc = inputVideo.get(CAP_PROP_FOURCC);
	double fps = inputVideo.get(CAP_PROP_FPS);
	Size frameSize (inputVideo.get(CAP_PROP_FRAME_WIDTH), inputVideo.get(CAP_PROP_FRAME_HEIGHT));
	VideoWriter outputVideo(outputFile, fourcc, fps, frameSize, true);
	if (!outputVideo.isOpened()) {
		printf("Could not open the output video for write: %s\n", outputFile.c_str());
		return;
	}

	// Process the video, frame by frame
	GpuTimer timer;
	timer.Start();
	Mat frame;
	while (inputVideo.read(frame)) {

		// copy the input image from CPU to GPU memory
		cv::cuda::GpuMat gpuFrame = cv::cuda::GpuMat(frame);
		cv::cuda::GpuMat gpuGradientFrame;

		processImage(gpuFrame, gpuGradientFrame); // extract grayscale gradient from the color frame
		cv::cuda::cvtColor(gpuGradientFrame, gpuFrame, COLOR_GRAY2BGR); // convert grayscale gradient to color

		// copy the result gradient from GPU to CPU and release GPU memory
		gpuFrame.download(frame);

		// release memory from GPU
		gpuGradientFrame.release();
		gpuFrame.release();

		// save current frame into the video
		outputVideo.write(frame);
	}
	timer.Stop();

	printf("Method processVideo() ran in: %f msecs, video size: %ux%u, total frames: %f, msecs/pixel: %f .\n",
			timer.Elapsed(), frameSize.width, frameSize.height, inputVideo.get(CAP_PROP_FRAME_COUNT),
			timer.Elapsed()/(frameSize.height*frameSize.width*inputVideo.get(CAP_PROP_FRAME_COUNT)));

	outputVideo.release();
	inputVideo.release();
}

}

